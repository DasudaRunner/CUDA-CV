#include "hip/hip_runtime.h"
#include "cudaImg.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <opencv2/opencv.hpp>


__global__ void getHist(unsigned char* dataIn, unsigned int* hist)
{

    int xdx = threadIdx.x + __umul24(blockIdx.x, blockDim.x);
    int ydx = threadIdx.y + __umul24(blockIdx.y, blockDim.y);

    int tid = xdx + ydx*gridDim.x*blockDim.x;

    if(tid < 256)
    {
        hist[tid]=0;
    }
    __syncthreads();
    atomicAdd(&hist[dataIn[tid]],1);

}

void getHist_gpu(unsigned char* dataIn,
                 unsigned int* hist,
                 dim3 tPerBlock,
                 dim3 bPerGrid)
{
    getHist<<<bPerGrid,tPerBlock>>>(dataIn,hist);
}